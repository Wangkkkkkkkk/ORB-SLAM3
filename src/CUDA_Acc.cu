#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <math.h>
#include "opencv2/opencv.hpp"
#include "opencv2/core/cuda.hpp"
#include "opencv2/core/cuda_stream_accessor.hpp"
#include <opencv2/features2d/features2d.hpp>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "ORBextractor.h"

using namespace std;

int indexs;

__device__ int nlevels;
__device__ float scaleFactor;
__device__ int isGetImageSize;

thrust::device_vector<uchar1*> d_srcs;
thrust::device_vector<uchar1*> d_dsts;

// __device__ ushort2* Keys;

inline __device__ __host__ int divUp(int A, int B) { return (A + B - 1) / B; }

// 灰度图像高斯滤波
__global__ void gray_gaussian_filtering(uchar1* img, uchar1* dst, int width, int height)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x + 16;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y + 16;

    // uchar v = img[idy * width + idx].x; // 读取判定点的灰度值
    // printf("idx: %d idy: %d width: %d\n", idx, idy, width);
    // printf("v: %d \n", v);

    if(idx < width && idy < height)
    {
        float v1 = img[idy * width + idx].x * 0.2725;
        float v2 = (img[(idy+1)*width + idx+1].x + img[(idy+1)*width + idx-1].x + img[(idy-1)*width + idx+1].x + img[(idy-1)*width + idx-1].x) * 0.0571;
        float v3 = (img[(idy+1)*width + idx].x + img[(idy-1)*width + idx].x + img[idy*width + idx+1].x + img[idy*width + idx-1].x) * 0.1248;
        dst[idy*width + idx].x = v1 + v2 + v3;
    }

    // if (idx < src.cols && idy < src.rows && src.at<int>(idy, idx) != 0)
    // {
    //     float v1 = src.at<int>(idy, idx) * 0.2725;
    //     float v2 = (src.at<int>(idy + 1, idx + 1) + src.at<int>(idy + 1, idx - 1) + src.at<int>(idy - 1, idx + 1) + src.at<int>(idy - 1, idx - 1)) * 0.0571;
    //     float v3 = (src.at<int>(idy + 1, idx) + src.at<int>(idy - 1, idx) + src.at<int>(idy, idx + 1) + src.at<int>(idy, idx - 1)) * 0.1248;
    //     dst.at<int>(idy, idx) = v1 + v2 + v3;
    // }
}

// 灰度图像高斯滤波 shared内存不能用于图像，会超出共享内存最大值
// __global__ void gray_gaussian_filtering_shared(uchar1* src, uchar1* dst, int width,int height)

// {
//     const int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     const int idy = blockIdx.y * blockDim.y + threadIdx.y;

//     __shared__ int tile[height][width];
//     tile[idy][idx] = src[idy * width + idx].x;
//     __syncthreads();

//     if(idx < width && idy < height)
//     {
//         float v1 = tile[idy][idx] * 0.2725;
//         float v2 = (tile[idy+1][idx+1] + tile[idy+1][idx-1] + tile[idy-1][idx+1] + tile[idy-1][idx-1]) * 0.0571;
//         float v3 = (tile[idy+1][idx] + tile[idy-1][idx] + tile[idy][idx+1] + tile[idy][idx-1]) * 0.1248;
//         dst[idy*width + idx].x = v1 + v2 + v3;
//     }
// }

// 使用类Fast 9-16 角点计算方法,检查该点是否通过 FAST 的筛选标准
__device__ bool fast_check(uchar1* img, const short idx, const short idy, const float radius, const int threshold, int width, int height)
{
    uchar v = img[idy * width + idx].x; // 读取判定点的灰度值
    // printf("idx: %d idy: %d width: %d\n", idx, idy, width);
    // printf("v: %d \n", v);
    uchar c[4];
    uchar drop_cnt = 0;
    for (uchar i = 0; i < 4; i++)
    {
        uchar values[4] = {0};
        float theta = M_PI_2 / 4 * i;
        uchar ox = radius * sin(theta);
        uchar oy = radius * cos(theta);

        c[0] = img[(idy - oy) * width + idx + ox].x;
        c[2] = img[(idy + oy) * width + idx - ox].x;

        values[0] = abs(c[0] - v) < threshold ? 1 : 0;
        values[1] = abs(c[2] - v) < threshold ? 1 : 0;
        if (values[0] & values[1])
            return false;

        c[1] = img[(idy + oy) * width + idx + ox].x;
        c[3] = img[(idy - oy) * width + idx - ox].x;
        values[2] = abs(c[1] - v) < threshold ? 1 : 0;
        values[3] = abs(c[3] - v) < threshold ? 1 : 0;
        if (values[2] & values[3])
            return false;

        drop_cnt += values[0] + values[1] + values[2] + values[3];
        if (drop_cnt >= 9)
            return false;
    }
    return true;
}

__device__ int fast_detect_16(uchar1* img, const short idx, const short idy, const int threshold, int width, int height)
{
    uchar v = img[idy * width + idx].x; // 读取判定点的灰度值

    int radiu = 3;
    uchar c[16];
    
    c[0] = img[(idy - radiu) * width + idx].x;
    c[4] = img[idy * width + idx + radiu].x;
    c[8] = img[(idy + radiu) * width + idx].x;
    c[12] = img[idy * width + idx - radiu].x;

    uchar num = 0;
    for (int i=0;i<4;i++) {
        uchar ind = i * 4;
        uchar values = abs(c[ind] - v);
        if (values > threshold) {
            num++;
        }
    }
    if (num < 3) {
        return -1;
    }

    c[1] = img[(idy - radiu) * width + idx + radiu - 2].x;
    c[2] = img[(idy - radiu + 1) * width + idx + radiu - 1].x;
    c[3] = img[(idy - radiu + 2) * width + idx + radiu].x;
    c[5] = img[(idy + radiu - 2) * width + idx + radiu].x;
    c[6] = img[(idy + radiu - 1) * width + idx + radiu - 1].x;
    c[7] = img[(idy + radiu) * width + idx + radiu - 2].x;
    c[9] = img[(idy + radiu) * width + idx - radiu + 2].x;
    c[10] = img[(idy + radiu - 1) * width + idx - radiu + 1].x;
    c[11] = img[(idy + radiu - 2) * width + idx - radiu].x;
    c[13] = img[(idy - radiu + 2) * width + idx - radiu].x;
    c[14] = img[(idy - radiu + 1) * width + idx - radiu + 1].x;
    c[15] = img[(idy - radiu) * width + idx - radiu + 2].x;

    num = 0;
    int response = 0;
    for (int i=0;i<16;i++) {
        uchar values = abs(c[i] - v);
        if (values > threshold) {
            num++;
        }
        response += values;
    }
    if (num >= 12) {
        return response;
    }
    else {
        return -1;
    }
}

// Harris Response计算
#define HARRIES_RADIUS 3
#define GAUSSIAN_SIGMA2 0.64f
#define CONSTANT_K 0.05f
__device__ float harris_response(uchar1* img, const int idx, const int idy, const float scale, int width, int height)
{
    // X, Y方向的梯度平方
    double A = 0, B = 0, C = 0;
    for (int v = -HARRIES_RADIUS; v <= HARRIES_RADIUS; v++)
        for (int u = -HARRIES_RADIUS; u <= HARRIES_RADIUS; u++)
        {
            int nx = idx + u * scale, ny = idy + v * scale;
            int scalei = int(scale);
            int Ix = (img[ny * width + nx + scalei].x - img[ny * width + nx - scalei].x) * 2 +
                        (img[(ny - scalei) * width + nx + scalei].x - img[(ny - scalei) * width + nx - scalei].x) +
                        (img[(ny + scalei) * width + nx + scalei].x - img[(ny + scalei) * width + nx - scalei].x);

            int Iy = (img[(ny + scalei) * width + nx].x - img[(ny - scalei) * width + nx].x) * 2 +
                        (img[(ny + scalei) * width + nx - scalei].x - img[(ny - scalei) * width + nx - scalei].x) +
                        (img[(ny + scalei) * width + nx + scalei].x - img[(ny - scalei) * width + nx + scalei].x);

            double gaussian_weight = exp(-(u * u + v * v) / (2 * 0.64));
            A += gaussian_weight * Ix * Ix;
            B += gaussian_weight * Iy * Iy;
            C += gaussian_weight * Ix * Iy;
        }
    double det_m = A * B - C * C;
    double trace_m = A + B;
    float score = det_m - CONSTANT_K * trace_m * trace_m;
    return score * 1e-9;
}

__global__ void fast(uchar1* img, const int threshold, int *dev_counter,
                        ushort3 *kptsLoc2D, uchar1* score_mat,
                        int width, int height) 
{
    const ushort idx = blockIdx.x * blockDim.x + threadIdx.x + 16;
    const ushort idy = blockIdx.y * blockDim.y + threadIdx.y + 16;

    if (idx < width - 16 && idy < height - 16)
    {
        // 检查是否通过 Fast 角点的收录标准
        int response = fast_detect_16(img, idx, idy, threshold, width, height);
        if (response > 0)
        {
            int pIdx = atomicAdd(dev_counter, 1);
            if (pIdx < 10000)
            {
                kptsLoc2D[pIdx] = {idx, idy, response};
                score_mat[idy * width + idx].x = response;
            }
        }
    }
}

__global__ void fast_grid(uchar1* img, int iniY, int maxY, int iniX, int maxX,
                            const int threshold, int *dev_counter,
                            ushort3 *kpts, uchar1* score_mat,
                            int width, int height) 
{
    const ushort idx = blockIdx.x * blockDim.x + threadIdx.x + iniX;
    const ushort idy = blockIdx.y * blockDim.y + threadIdx.y + iniY;

    if (idx < maxX && idy < maxY)
    {
        // 检查是否通过 Fast 角点的收录标准
        int response = fast_detect_16(img, idx, idy, threshold, width, height);
        if (response > 0)
        {
            int pIdx = atomicAdd(dev_counter, 1);
            if (pIdx < 50000)
            {
                kpts[pIdx] = {idx, idy, response};
                score_mat[idy * width + idx].x = response;
            }
        }
    }
}

#define NM_RADIUS 3 // 非极大抑制范围 Radius
// 非极大抑制，剔除部分 Edge 点(negtive)，同时保存特征点的三维位置信息
__global__ void nonmaxSuppression(ushort3* kptsLoc2D, int kpt_num,
                                  uchar1* scoreMat, int* dev_counter,
                                  ushort3* frame_kpts,
                                  int width, int height)
{
    const short kpIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (kpIdx < kpt_num)
    {
        ushort3 loc = kptsLoc2D[kpIdx];
        int score = scoreMat[loc.y * width + loc.x].x;

        for (int i = -NM_RADIUS; i <= NM_RADIUS; i++)
            for (int j = -NM_RADIUS; j <= NM_RADIUS; j++)
                if (score < scoreMat[(loc.y + i) * width + loc.x + j].x)
                    return;

        int idx = atomicAdd(dev_counter, 1);
        // 写入特征点图像坐标，三维空间坐标
        if (idx < 50000)
        {
            frame_kpts[idx] = loc;
        }
    }
}

// CUDA 初始化，主要得到 金字塔层数 以及 是否得到图像尺寸设为 0
void ORB_SLAM3::ORBextractor::CUDA_Initial(int _nlevel, float _scaleFactor)
{
    indexs = 0;

    nlevels = _nlevel;
    scaleFactor = _scaleFactor;
    isGetImageSize = 0;

    d_srcs.resize(nlevels);
    d_dsts.resize(nlevels);

}

int ORB_SLAM3::ORBextractor::getIndex()
{
    return indexs;
}

// 创建GPU内存空间
void ORB_SLAM3::ORBextractor::getPyramid(cv::InputArray image, int level)
{

/*
    level:0 rows: 480 cols: 752
    level:1 rows: 400 cols: 627
    level:2 rows: 333 cols: 522
    level:3 rows: 278 cols: 435
    level:4 rows: 231 cols: 363
    level:5 rows: 193 cols: 302
    level:6 rows: 161 cols: 252
    level:7 rows: 134 cols: 210
*/
    cv::Mat src = image.getMat();
    int height = src.rows;
    int width = src.cols;

    // GPU内存初始化
    if (isGetImageSize == 0) {
        uchar1* d_src;
        uchar1* d_dst;

        int rows = height;
        int cols = width;
        // cout<< "rows: " << rows << " cols: " << cols <<endl;

        size_t memSize = cols * rows * sizeof(uchar1);
        hipMalloc((void**)&d_src, memSize);
        hipMalloc((void**)&d_dst, memSize);

        d_srcs[level] = d_src;
        d_dsts[level] = d_dst;

        // 只创建一次内存空间
        if (level == nlevels-1) {
            isGetImageSize = 1;

            // size_t memSizes = 5000 * sizeof(ushort2);
            // hipMalloc((void**)&Keys, memSizes);
        }
    }

    // 图像信息赋值
    // size_t memSize = width * height * sizeof(uchar1);
    // hipMemcpy(d_srcs[level], src.data, memSize, hipMemcpyHostToDevice);

}

void ORB_SLAM3::ORBextractor::GBandCD_CUDA(cv::InputArray image, int level)
{
    indexs++;

    cv::Mat src = image.getMat();
    // cout<< "src[16][16]: " << int(src.at<uchar>(16, 16)) <<endl;

    size_t memSize = src.cols * src.rows * sizeof(uchar1);

    hipMemcpy(d_srcs[level], src.data, memSize, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid(divUp(src.cols, block.x),
              divUp(src.rows, block.y));
    // dim3 block(1, 1);
    // dim3 grid(1, 1);
    gray_gaussian_filtering<<<grid, block>>>(d_srcs[level], d_dsts[level], src.cols, src.rows);

    hipMemcpy(src.data, d_dsts[level], memSize, hipMemcpyDeviceToHost);

    // hipFree(d_src);
    // hipFree(d_dst);
}

void ORB_SLAM3::ORBextractor::ExtractorPoint(cv::InputArray image, int level, vector<cv::KeyPoint> &vKeys)
{
    cv::Mat src = image.getMat();
    int Rows = src.rows;
    int Cols = src.cols;

    // size_t memSize = src.cols * src.rows * sizeof(uchar1);
    // uchar1* d_src;
    // hipMalloc((void**)&d_src, memSize);
    // hipMemcpy(d_src, src.data, memSize, hipMemcpyHostToDevice);

    int iniThFAST = 25;
    int *dev_counter;
    hipMalloc((void**)&dev_counter, sizeof(int));
    hipMemset(dev_counter, 0, sizeof(int));

    ushort3* Keys;
    size_t memSizes = 50000 * sizeof(ushort3);
    hipMalloc((void**)&Keys, memSizes);

    size_t memSize = Cols * Rows * sizeof(uchar1);
    hipMemcpy(d_srcs[level], src.data, memSize, hipMemcpyHostToDevice);

    uchar1* score_mat;
    hipMalloc((void**)&score_mat, memSize);
    hipMemset(score_mat, 0, memSize);

    dim3 block(16, 16);
    dim3 grid(divUp(Cols - 2*16, block.x),
              divUp(Rows - 2*16, block.y));
    fast<<<grid, block>>>(d_srcs[level], iniThFAST, dev_counter, Keys, score_mat, Cols, Rows);

    int* n;
    n = (int*)malloc(sizeof(int));
    hipMemcpy(n, dev_counter, sizeof(int), hipMemcpyDeviceToHost);
    cout<< "fast KeyPoint num: " << *n <<endl;
    *n = min(*n, 50000);

    int *kp_counter;
    hipMalloc((void**)&kp_counter, sizeof(int));
    hipMemset(kp_counter, 0, sizeof(int));

    ushort3* KeyPoints;
    hipMalloc((void**)&KeyPoints, memSizes);

    dim3 block_nm(64);
    dim3 grid_nm(divUp(*n, block_nm.x));
    nonmaxSuppression<<<grid_nm, block_nm>>>(Keys, *n, score_mat, kp_counter, KeyPoints, Cols, Rows);

    int* num;
    num = (int*)malloc(sizeof(int));
    hipMemcpy(num, kp_counter, sizeof(int), hipMemcpyDeviceToHost);
    cout<< "nonmax KeyPoint num: " << *num <<endl;

    *num = min(*num, 50000);
    ushort3* kptsLoc = (ushort3*)malloc(*num * sizeof(ushort3));
    hipMemcpy(kptsLoc, KeyPoints, *num * sizeof(ushort3), hipMemcpyDeviceToHost);

    for (int pidx = 0; pidx < *num; pidx++)
    {
        cv::KeyPoint kp;
        kp.pt.x = kptsLoc[pidx].x - 16;
        kp.pt.y = kptsLoc[pidx].y - 16;
        kp.response = kptsLoc[pidx].z;
        kp.octave = level;
        kp.size = 0;
        vKeys.push_back(kp);

        // cv::Scalar color(0, 255, 0);
        // cv::circle(src, pt, 3, color);
    }

    // cv::imshow("color coarse fast ", src);
    // cv::waitKey();

    // hipDeviceSynchronize();

    hipFree(dev_counter);
    hipFree(kp_counter);
    hipFree(Keys);
    hipFree(score_mat);
    hipFree(KeyPoints);

    free(kptsLoc);
    free(n);
    free(num);
}

void ORB_SLAM3::ORBextractor::ExtractorPointGrid(cv::InputArray image, int level, vector<cv::KeyPoint> &vKeys,
                                                int minBorderX, int minBorderY, int maxBorderX, int maxBorderY,
                                                int nCols, int nRows, int wCell, int hCell)
{
    cv::Mat src = image.getMat();

    int iniThFAST = 50;
    int *dev_counter;
    hipMalloc((void**)&dev_counter, sizeof(int));
    hipMemset(dev_counter, 0, sizeof(int));

    ushort3* Keys;
    size_t memSizes = 10000 * sizeof(ushort3);
    hipMalloc((void**)&Keys, memSizes);

    size_t memSize = src.cols * src.rows * sizeof(uchar1);
    hipMemcpy(d_srcs[level], src.data, memSize, hipMemcpyHostToDevice);

    uchar1* score_mat;
    hipMalloc((void**)&score_mat, memSize);
    hipMemset(score_mat, 0, memSize);

    // hipStream_t stream[nRows * nCols];

    // for (int i=0;i<nRows;i++) {
    //     for (int j=0;j<nCols;j++) {
    //         hipStreamCreate(&stream[i * nRows + j]);
    //     }
    // }

    //开始遍历图像网格，还是以行开始遍历的
    for(int i=0; i<nRows; i++)
    {
        //计算当前网格初始行坐标
        const int iniY =minBorderY+i*hCell;
        //计算当前网格最大的行坐标，这里的+6=+3+3，即考虑到了多出来3是为了cell边界像素进行FAST特征点提取用
        //前面的EDGE_THRESHOLD指的应该是提取后的特征点所在的边界，所以minBorderY是考虑了计算半径时候的图像边界
        //目测一个图像网格的大小是25*25啊
        int maxY = iniY+hCell+6;

        //如果初始的行坐标就已经超过了有效的图像边界了，这里的“有效图像”是指原始的、可以提取FAST特征点的图像区域
        if(iniY>=maxBorderY-3)
            //那么就跳过这一行
            continue;
        //如果图像的大小导致不能够正好划分出来整齐的图像网格，那么就要委屈最后一行了
        if(maxY>maxBorderY)
            maxY = maxBorderY;

        //开始列的遍历
        for(int j=0; j<nCols; j++)
        {
            //计算初始的列坐标
            const int iniX =minBorderX+j*wCell;
            //计算这列网格的最大列坐标，+6的含义和前面相同
            int maxX = iniX+wCell+6;

            //判断坐标是否在图像中
            if(iniX>=maxBorderX-3)
                continue;

            //如果最大坐标越界那么委屈一下
            if(maxX>maxBorderX)
                maxX = maxBorderX;

            dim3 block(16, 16);
            dim3 grid(divUp(maxX - iniX, block.x),
                    divUp(maxY - iniY, block.y));
            
            fast_grid<<<grid, block>>>(d_srcs[level], iniY, maxY, iniX, maxX,
                                        iniThFAST, dev_counter, Keys, score_mat,
                                        src.cols, src.rows);

        }
    }

    int* n;
    n = (int*)malloc(sizeof(int));
    hipMemcpy(n, dev_counter, sizeof(int), hipMemcpyDeviceToHost);
    // cout<< "KeyPoint num: " << *n <<endl;



    ushort2* kptsLoc = (ushort2*)malloc(*n * sizeof(ushort2));
    hipMemcpy(kptsLoc, Keys, *n * sizeof(ushort2), hipMemcpyDeviceToHost);

    // for (int pidx = 0; pidx < *n; pidx++)
    // {
    //     static cv::Point2i pt;
    //     pt.x = kptsLoc[pidx].x;
    //     pt.y = kptsLoc[pidx].y;
    //     // cout<< "x: " << pt.x << " y: " << pt.y <<endl;
    //     cv::Scalar color(0, 255, 0);
    //     cv::circle(src, pt, 3, color);
    // }

    // cv::imshow("color coarse fast ", src);
    // cv::waitKey();

    // for (int i=0;i<nRows;i++) {
    //     for (int j=0;j<nCols;j++) {
    //         hipStreamDestroy(stream[i * nRows + j]);
    //     }
    // }

    hipFree(dev_counter);
    hipFree(Keys);
    hipFree(score_mat);

    free(n);
    free(kptsLoc);
}

// 释放内存空间
void ORB_SLAM3::ORBextractor::deleteMem()
{
    if (!d_srcs.empty()) {
        for (int i=0;i<nlevels;i++) {
            hipFree(d_srcs[i]);
            hipFree(d_dsts[i]);
        }
    }
}